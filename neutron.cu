#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <stdio.h>
#include <math.h>
#include <time.h>
#include <sys/time.h>
#include <hiprand/hiprand_kernel.h>

#include <ctime>
#define OUTPUT_FILE "/tmp/absorbed.dat"

char info[] = "\
Usage:\n\
    neutron-seq H Nb C_c C_s\n\
\n\
    H  : épaisseur de la plaque\n\
    Nb : nombre d'échantillons\n\
    C_c: composante absorbante\n\
    C_s: componente diffusante\n\
\n\
Exemple d'execution : \n\
    neutron-seq 1.0 500000000 0.5 0.5\n\
";

/* setup_kernel <<<NbBlocks,NbThreadsParBloc>>> (devStates,unsigned(time(NULL))); //initialisation de l'état curandState pour chaque thread
 * notre gettimeofday()
 */
double my_gettimeofday(){
  struct timeval tmp_time;
  gettimeofday(&tmp_time, NULL);
  return tmp_time.tv_sec + (tmp_time.tv_usec * 1.0e-6L);
}


__global__ void setup_kernel(hiprandState *state)
{
    int id = threadIdx.x + blockIdx.x * blockDim.x;
    /* Each thread gets same seed, a different sequence 
       number, no offset */
    hiprand_init(0, id, 0, &state[id]);
}


  //int id = threadIdx.x + blockIdx.x * blockDim.x;



__global__ void generate_kernel(hiprandState *state,
                                int step, unsigned int nbThread, dim3 TailleGrille, unsigned int N, float c, float c_c,float h, int* R, int* B,int* T,float* absorbed, int* j)
{
	int id = threadIdx.x + blockIdx.x *TailleGrille.x;

	/* Copy state to local memory for efficiency */
    hiprandState localState = state[id];
    /* Generate pseudo-random unsigned ints */
    float u;

    int r=0;
    int b=0;
    int t=0;

    __shared__ int Rtab[1024];
    __shared__ int Btab[1024];
    __shared__ int Ttab[1024];

    
    float d, x, L; 

  	int debut_for = id*step ;
  	int fin_for = (id+1)*step;

  	if(id == nbThread) fin_for = N;
  
    for (int i = debut_for; i < fin_for; i++) {
        d = 0.0;
        x = 0.0;

        while (1) {

            u = hiprand_uniform(&localState);
            L = -(1 / c) * log(u);
            x = x + L * cos(d);  //initialisation de l'état curandState pour chaque thread
            if (x < 0) {
                r++;
                break;
            } else if (x >= h) {
                t++;
                break;
            } else if ((u = hiprand_uniform(&localState)) < c_c / c) {
                b++;
                atomicAdd(j,1);
                absorbed[(*j)] = x;
                break;
            } else {
                u = hiprand_uniform(&localState);
                d = u * M_PI;
            }
        }
    }

    Rtab[threadIdx.x]=r;
    Btab[threadIdx.x]=b;
    Ttab[threadIdx.x]=t;

    //reduction :
    __syncthreads();
    int k=blockDim.x/2;
    while (k>0){
        if(threadIdx.x<k){
            Rtab[threadIdx.x]+=Rtab[threadIdx.x +k];
            Btab[threadIdx.x]+=Btab[threadIdx.x +k];
            Ttab[threadIdx.x]+=Ttab[threadIdx.x +k];
        }
        k/=2;
        __syncthreads();
    }
    if(threadIdx.x==0){
        atomicAdd(R,Rtab[0]);
        atomicAdd(B,Btab[0]);
        atomicAdd(T,Ttab[0]);
    }
 
  /* Copy state back to global memory 
  r=atomicAdd(R,r);
  b=atomicAdd(B,b);
  t=atomicAdd(T,t);
  */
  state[id] = localState;

}



int main(int argc, char *argv[]) {
    // La distance moyenne entre les interactions neutron/atome est 1/c. 
    // c_c et c_s sont les composantes absorbantes et diffusantes de c. 
    float c, c_c, c_s;
    // épaisseur de la plaque
    float h;
    // nombre d'échantillons
    int n;
    // nombre de neutrons refléchis, absorbés et transmis
    int r, b, t;
    // chronometrage
    double start, finish;
    int j = 0; // compteurs 

    if( argc == 1)
    fprintf( stderr, "%s\n", info);

    // valeurs par defaut
    h = 1.0;
    n = 500000000;
    c_c = 0.5;
    c_s = 0.5;

    // recuperation des parametres
    if (argc > 1)
    h = atof(argv[1]);
    if (argc > 2)
    n = atoi(argv[2]);
    if (argc > 3)
    c_c = atof(argv[3]);
    if (argc > 4)
    c_s = atof(argv[4]);
    r = b = t = 0;
    c = c_c + c_s;

    // affichage des parametres pour verificatrion
    printf("Épaisseur de la plaque : %4.g\n", h);
    printf("Nombre d'échantillons  : %d\n", n);
    printf("C_c : %g\n", c_c);
    printf("C_s : %g\n", c_s);


    float *absorbed;
    absorbed = (float *) calloc(n, sizeof(float));

      /* allocation de memoire GPU*/

    float *d_absorbed;
    int size = n*sizeof(float);
    hipMalloc((void**)&d_absorbed, size);

    int *d_r, *d_b, *d_t, *d_j;
    hipMalloc((void**)&d_r, sizeof(int));
    hipMalloc((void**)&d_b, sizeof(int));
    hipMalloc((void**)&d_t, sizeof(int));
    hipMalloc((void**)&d_j, sizeof(int));

    // Transfert CPU -> GPU
    hipMemcpy(d_absorbed, absorbed, size, hipMemcpyHostToDevice);
    hipMemcpy(d_r, &r, sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_b, &b, sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_t, &t, sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_j, &j, sizeof(int), hipMemcpyHostToDevice);


    // Definition nombre de threads
    dim3 TailleGrille, ThreadparBlock;

    ThreadparBlock.x = 1024; //32*32
    ThreadparBlock.y = 1;
    ThreadparBlock.z = 1;

    TailleGrille.x = 1024;
    TailleGrille.y = 1;
    TailleGrille.z = 1;


    int nbThread = TailleGrille.x*ThreadparBlock.x;

    int step = n/nbThread; //nb de neutrons gérés par chaque thread

    hiprandState *d_States;
    /* Allocation un vecteur d'etat par thread */
    hipMalloc((void **)&d_States, nbThread*sizeof(hiprandState));  

    
    // debut du chronometrage
    start = my_gettimeofday();

    //appel kernel1 : initialisation states
    setup_kernel<<<TailleGrille,ThreadparBlock>>>(d_States);

    //appel kernel2 : calcul
    generate_kernel<<<TailleGrille,ThreadparBlock>>>(d_States,step,nbThread,TailleGrille,n,c,c_c,h,d_r,d_b,d_t,d_absorbed, d_j);

    // fin du chronometrage
    finish = my_gettimeofday();

    // Transfert GPU-> CPU
    hipMemcpy(absorbed, d_absorbed, size, hipMemcpyDeviceToHost);
    hipMemcpy(&r, d_r, sizeof(int), hipMemcpyDeviceToHost);
    hipMemcpy(&b, d_b, sizeof(int), hipMemcpyDeviceToHost);
    hipMemcpy(&t, d_t, sizeof(int), hipMemcpyDeviceToHost);



    printf("\nPourcentage des neutrons refléchis : %4.2g\n", (float) r / (float) n);
    printf("Pourcentage des neutrons absorbés : %4.2g\n", (float) b / (float) n);
    printf("Pourcentage des neutrons transmis : %4.2g\n", (float) t / (float) n);

    printf("\nTemps total de calcul: %.8g sec\n", finish - start);
    printf("Millions de neutrons /s: %.2g\n", (double) n / ((finish - start)*1e6));

    // ouverture du fichier pour ecrire les positions des neutrons absorbés
    FILE *f_handle = fopen(OUTPUT_FILE, "w");
    if (!f_handle) {
    fprintf(stderr, "Cannot open " OUTPUT_FILE "\n");
    exit(EXIT_FAILURE);
    }

    for (j = 0; j < b; j++){
        fprintf(f_handle, "%f\n", absorbed[j]);
    }

    // fermeture du fichier
    fclose(f_handle);
    printf("Result written in " OUTPUT_FILE "\n"); 

    free(absorbed);
    hipFree(d_absorbed);
    hipFree(d_States);
    hipFree(d_r);
    hipFree(d_b);
    hipFree(d_t);

}
